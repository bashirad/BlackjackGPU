#include "hip/hip_runtime.h"
/*
Copyright (c) Ron Coleman 
Permission is hereby granted, free of charge, to any person obtaining
a copy of this software and associated documentation files (the
"Software"), to deal in the Software without restriction, including
without limitation the rights to use, copy, modify, merge, publish,
distribute, sublicense, and/or sell copies of the Software, and to
permit persons to whom the Software is furnished to do so, subject to
the following conditions:

The above copyright notice and this permission notice shall be
included in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "Card.h"
#include "Hand.h"
#include "Strategy.h"
#include "Player.h"
#include "Game.h"
#include "Helpers.h"
#include "Kpax.h"

#define ACE_AS_11 10

__device__ __shared__ hiprandState_t* states;

__device__ int random(int min, int max, hiprandState_t* state) {

	// See https://stackoverflow.com/questions/2509679/how-to-generate-a-random-integer-number-from-within-a-range/6852396
	return (hiprand(state) % (max + 1 - min)) + min;
}

///////////////// Card

__device__ Card Card_(Rank rank, Suit suit) {
	Card card = { rank, suit };
	return card;
}

__device__ Card Card_(int rank, Suit suit) {
	Card card = { (Rank)rank, suit };
	return card;
}

__device__ Suit randomSuit(hiprandState_t* state) {
	Suit suits[] = { HEARTS, SPADES, DIAMONDS, CLUBS };

	int index = random(0, 3, state);

	return suits[index];
}

__device__ Rank randomRank(hiprandState_t* state) {
	Rank rank = (Rank)random(ACE, KING, state);

	return rank;
}

__device__ Card deal() {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState_t* state = &states[index];

	// Get a random suit
	Suit suit = randomSuit(state);

	// Get a rank
	Rank rank = randomRank(state);

	// Make a card
	Card card = Card_(rank, suit);
	return card;
}

__device__ bool isFace(Card* card) {
	return card->rank == JACK || card->rank == QUEEN || card->rank == KING;
}

__device__ bool isAce(Card* card) {
	return card->rank == ACE;
}

__device__ bool is10(Card* card) {
	return isFace(card) || card->rank == TEN;
}


////////// Hand

__device__ Hand Hand_() {
	Hand hand = { {
			Card_(NO_RANK, NO_SUIT),
			Card_(NO_RANK, NO_SUIT),
			Card_(NO_RANK, NO_SUIT),
			Card_(NO_RANK, NO_SUIT),
			Card_(NO_RANK, NO_SUIT) },
			0, 0, 1.0, NULL };

	return hand;
}

__device__ Hand Hand_(void* player) {
	Hand hand = { {
			Card_(NO_RANK, NO_SUIT),
			Card_(NO_RANK, NO_SUIT),
			Card_(NO_RANK, NO_SUIT),
			Card_(NO_RANK, NO_SUIT),
			Card_(NO_RANK, NO_SUIT) },
			0, 0, 1.0, player };

	return hand;
}

__device__ Card hit(Player* player) {
	int sz = player->size;

	return hit(player, sz - 1);
}

__device__ Card hit(Player* player, int handno) {
	return hit(&player->hands[handno]);
}

__device__ int score(Hand* hand) {
	int sum = 0;

	int nAces = 0;

	for (int k = 0; k < hand->size; k++) {
		Card card = hand->cards[k];
		sum += isFace(&hand->cards[k]) ? 10 : hand->cards[k].rank;

		// If rank == ACE_AS_11 won't be counted here (correctly!)
		if (card.rank == ACE)
			nAces++;
	}

	// Account for soft aces
	for (int k = 0; k < nAces; k++) {
		if (sum + ACE_AS_11 > 21)
			break;

		sum += ACE_AS_11;
	}

	return sum;
}

__device__ bool isBroke(Hand* hand) {
	return hand->value > 21;
}

__device__ bool isCharlie(Hand* hand) {
	return hand->size == 5 && hand->value <= 21;
}

__device__ bool isBlackjack(Hand* hand) {
	return hand->size == 2 && hand->value == 21;
}

__device__ bool isBlackjack(void* player, Hand* hand) {
	Player* playor = (Player*)player;

	return isBlackjack(hand) && playor->size == 1;
}

__device__ Card hit(Hand* hand) {
	Card card = deal();

	hit(hand, &card);

	return card;
}

__device__ void hit(Hand* hand, Card* card) {
	int index = hand->size++;

	hand->cards[index] = *card;

	hand->value = score(hand);

	return;
}

__device__ bool isPair(Hand* hand) {
	if (hand->size != 2)
		return false;

	Card card1 = hand->cards[0];
	Card card2 = hand->cards[1];

	return card1.rank == card2.rank;
}

__device__ bool isAcePlusX(Hand* hand) {
	if (hand->size != 2)
		return false;

	Card card1 = hand->cards[0];
	Card card2 = hand->cards[1];

	return (card1.rank == ACE && card2.rank != ACE) || (card1.rank != ACE && card2.rank == ACE);
}

/////////////// Strategy

__host__ __device__ Strategy Strategy_() {
	Strategy s = {
		0.0,
		{
			//          2  3  4  5  6  7  8  9  T  A
			/*21*/		X, X, X, X, X, X, X, X, X, X, // Section I: lower=0, upper=9
			/*20*/		X, X, X, X, X, X, X, X, X, X,
			/*19*/		X, X, X, X, X, X, X, X, X, X,
			/*18*/		X, X, X, X, X, X, X, X, X, X,
			/*17*/		X, X, X, X, X, X, X, X, X, X,
			/*16*/		X, X, X, X, X, X, X, X, X, X,
			/*15*/		X, X, X, X, X, X, X, X, X, X,
			/*14*/		X, X, X, X, X, X, X, X, X, X,
			/*13*/		X, X, X, X, X, X, X, X, X, X,
			/*12*/		X, X, X, X, X, X, X, X, X, X,

			/*11*/		X, X, X, X, X, X, X, X, X, X, // Section II: lower=10, upper=17
			/*10*/		X, X, X, X, X, X, X, X, X, X,
			/*9 */		X, X, X, X, X, X, X, X, X, X,
			/*8 */		X, X, X, X, X, X, X, X, X, X,
			/*7 */		X, X, X, X, X, X, X, X, X, X,
			/*6 */		X, X, X, X, X, X, X, X, X, X,
			/*5 */		X, X, X, X, X, X, X, X, X, X,
			/*4 */		X, X, X, X, X, X, X, X, X, X, // When number of splits limited by house use this row

			/*A,K */	X, X, X, X, X, X, X, X, X, X, // Section III: lower=18, upper=29
			/*A,Q */	X, X, X, X, X, X, X, X, X, X,
			/*A,J */	X, X, X, X, X, X, X, X, X, X,
			/*A,T */	X, X, X, X, X, X, X, X, X, X,
			/*A,9 */	X, X, X, X, X, X, X, X, X, X,
			/*A,8 */	X, X, X, X, X, X, X, X, X, X,
			/*A,7 */	X, X, X, X, X, X, X, X, X, X,
			/*A,6 */	X, X, X, X, X, X, X, X, X, X,
			/*A,5 */	X, X, X, X, X, X, X, X, X, X,
			/*A,4 */	X, X, X, X, X, X, X, X, X, X,
			/*A,3 */	X, X, X, X, X, X, X, X, X, X,
			/*A,2 */	X, X, X, X, X, X, X, X, X, X,

			/*A,A */	X, X, X, X, X, X, X, X, X, X, // Section IV: lower=30, upper=42
			/*T,K */	X, X, X, X, X, X, X, X, X, X,
			/*T,Q */	X, X, X, X, X, X, X, X, X, X,
			/*T,J */	X, X, X, X, X, X, X, X, X, X,
			/*T,T */	X, X, X, X, X, X, X, X, X, X,
			/*9,9 */	X, X, X, X, X, X, X, X, X, X,
			/*8,8 */	X, X, X, X, X, X, X, X, X, X,
			/*7,7 */	X, X, X, X, X, X, X, X, X, X,
			/*6,6 */	X, X, X, X, X, X, X, X, X, X,
			/*5,5 */	X, X, X, X, X, X, X, X, X, X,
			/*4,4 */	X, X, X, X, X, X, X, X, X, X,
			/*3,3 */	X, X, X, X, X, X, X, X, X, X,
			/*2,2 */	X, X, X, X, X, X, X, X, X, X,
		},
		{ { 0,  9 },
		{ 10, 17 },
		{ 18, 29 },
		{ 30, 42 } },
		{0, 0, 0, 0, 0}
	};
	return s;
}

__host__ __device__ Strategy BasicStrategy_() {
	Strategy s = {
		0.0,
		{
			//          2  3  4  5  6  7  8  9  T  A
			/*21*/		S, S, S, S, S, S, S, S, S, S, // Section I: lower=0, upper=9
			/*20*/		S, S, S, S, S, S, S, S, S, S,
			/*19*/		S, S, S, S, S, S, S, S, S, S,
			/*18*/		S, S, S, S, S, S, S, S, S, S,
			/*17*/		S, S, S, S, S, S, S, S, S, S,
			/*16*/		S, S, S, S, S, H, H, H, H, H,
			/*15*/		S, S, S, S, S, H, H, H, H, H,
			/*14*/		S, S, S, S, S, H, H, H, H, H,
			/*13*/		S, S, S, S, S, H, H, H, H, H,
			/*12*/		H, H, S, S, S, H, H, H, H, H,

			/*11*/		D, D, D, D, D, D, D, D, D, H, // Section II: lower=10, upper=17
			/*10*/		D, D, D, D, D, D, D, D, H, H,
			/*9 */		H, D, D, D, D, H, H, H, H, H,
			/*8 */		H, H, H, H, H, H, H, H, H, H,
			/*7 */		H, H, H, H, H, H, H, H, H, H,
			/*6 */		H, H, H, H, H, H, H, H, H, H,
			/*5 */		H, H, H, H, H, H, H, H, H, H,
			/*4 */		H, H, H, H, H, H, H, H, H, H, // When number of splits limited by house use this row

			/*A,K */	S, S, S, S, S, S, S, S, S, S, // Section III: lower=18, upper=29
			/*A,Q */	S, S, S, S, S, S, S, S, S, S,
			/*A,J */	S, S, S, S, S, S, S, S, S, S,
			/*A,T */	S, S, S, S, S, S, S, S, S, S,
			/*A,9 */	S, S, S, S, S, S, S, S, S, S,
			/*A,8 */	S, S, S, S, S, S, S, S, S, S,
			/*A,7 */	S, D, D, D, D, S, S, H, H, H,
			/*A,6 */	H, D, D, D, D, H, H, H, H, H,
			/*A,5 */	H, H, D, D, D, H, H, H, H, H,
			/*A,4 */	H, H, D, D, D, H, H, H, H, H,
			/*A,3 */	H, H, H, D, D, H, H, H, H, H,
			/*A,2 */	H, H, H, D, D, H, H, H, H, H,

			/*A,A */	P, P, P, P, P, P, P, P, P, P, // Section IV: lower=30, upper=42
			/*T,K */	S, S, S, S, S, S, S, S, S, S,
			/*Q,Q */	S, S, S, S, S, S, S, S, S, S,
			/*J,J */	S, S, S, S, S, S, S, S, S, S,
			/*T,T */	S, S, S, S, S, S, S, S, S, S,
			/*9,9 */	P, P, P, P, P, S, P, P, S, S,
			/*8,8 */	P, P, P, P, P, P, P, P, P, P,
			/*7,7 */	P, P, P, P, P, P, H, H, H, H,
			/*6,6 */	P, P, P, P, P, H, H, H, H, H,
			/*5,5 */	D, D, D, D, D, D, D, D, H, H,
			/*4,4 */	H, H, H, P, P, H, H, H, H, H,
			/*3,3 */	P, P, P, P, P, P, H, H, H, H,
			/*2,2 */	P, P, P, P, P, P, H, H, H, H,
		},
		{ { 0,   9 },
		{ 10, 17 },
		{ 18, 29 },
		{ 30, 42 } },
		{ 0, 0, 0, 0, 0 }
	};
	return s;
}

#define RANK(rank) (rank >= 10 ? 10 : rank)

__device__ Play doSection4(Hand* hand, Card* upcard, Strategy* strategy) {
	Card card1 = hand->cards[0];

	int offset = 0;

	if (!isAce(&card1))
		offset = KING - card1.rank + 1;

	int row = strategy->jumpTab[SECTION4].lower + offset;

	int col = isAce(upcard) ? 9 : RANK(upcard->rank) - 2;

	int index = row * 10 + col;

	Play play = strategy->rules[index];

	return play;
}

__device__ Play doSection3(Hand* hand, Card* upcard, Strategy* strategy) {
	Card card1 = hand->cards[0];
	Card card2 = hand->cards[1];

	Card card = card1;
	if (card1.rank == ACE)
		card = card2;

	int offset = KING - card.rank;

	int row = strategy->jumpTab[SECTION3].lower + offset;

	int col = isAce(upcard) ? 9 : RANK(upcard->rank) - 2;

	int index = row * 10 + col;

	Play play = strategy->rules[index];

	return play;
}

__device__ Play doSection2(Hand* hand, Card* upcard, Strategy* strategy) {
	int offset = 11 - hand->value;

	int row = strategy->jumpTab[SECTION2].lower + offset;

	int col = isAce(upcard) ? 9 : RANK(upcard->rank) - 2;

	int index = row * 10 + col;

	Play play = strategy->rules[index];

	// Double-down when more than 2 cards means hit -- see Instructional Services (1983, 2000)
	if (play == DOUBLE_DOWN && hand->size > 2)
		return HIT;

	return play;
}

__device__ Play doSection1(Hand* hand, Card* upcard, Strategy* strategy) {
	int offset = 21 - hand->value;

	int row = strategy->jumpTab[SECTION1].lower + offset;

	int col = isAce(upcard) ? 9 : RANK(upcard->rank) - 2;

	int index = row * 10 + col;

	Play play = strategy->rules[index];

	// Double-down when more than 2 cards means hit -- see Instructional Services (1983, 2000)
	if (play == DOUBLE_DOWN && hand->size > 2)
		return HIT;

	return play;
}


/////////////// Player

__device__ Player Player_(Strategy* strategy) {
	// TODO: the player's hand is not pointing back to the player
	Player player = { { Hand_() }, 1, strategy, 0.0 };

	return player;
}

__device__ void init(Player* player) {
	player->hands[0].player = player;
}

__device__ int add(Player* player, Hand* hand) {
	int index = player->size++;

	hand->player = player;

	player->hands[index] = *hand;

	return index;
}


/////////////// Game

__host__ Game Game_() {
	return{ { 0, 0, 0, 0, 0, 0, 0 }, 0, 0.0 };
}

__device__ void play(Strategy* strategy, Game* statistics) {
	// Create the heads-up game
	Player player = Player_(strategy);
	init(&player);

	Hand dealer = Hand_();

	// Deal the initial round
	hit(&player);

	hit(&dealer);

	hit(&player);

	hit(&dealer);

	Card upcard = dealer.cards[0];
	
	// Play the player's hand
	play(&player, &upcard);

	// Player the dealer's hand
	play(&dealer, &player, statistics);

	// Save the statistics.
	statistics->pl += player.pl;

	statistics->nohands += player.size;

//	int sum = 0;
//
//	int n = sizeof(statistics->count) / sizeof(int);
//
//	for (int index = 0; index < n; index++)
//		sum += statistics->count[index];
}

__device__ void play(Player* player, Card* upcard) {
	playout(&player->hands[0], upcard);
}

__device__ void playout(Hand* hand, Card* upcard) {
	Play play = getPlay(hand, upcard);

	Player* player = (Player*)hand->player;
	player->strategy->visits[play]++;

	switch (play) {
	case NO_PLAY:
		break;

	case STAY:
		break;

	case HIT:
		hit(hand);

		if (isBroke(hand) || isBlackjack(hand))
			return;

		if (hand->size >= MAX_HAND_CARDS)
			return;

		///*
		//if (isCharlie(hand))
		//return;
		//*/

		playout(hand, upcard);
		break;

	case DOUBLE_DOWN:
		// Double bet and hit once
		hand->bet *= 2.0;

		hit(hand);
		break;

	case SPLIT:
		if (hand->size != 2)
			return;

		split(hand, upcard);
		break;
	}

	return;
}

__device__ void split(Hand* hand1, Card* upcard) {
	// If there's a split overflow, fallback to non-split option
	Player* player = (Player*)hand1->player;

	if (player->size >= MAX_YOUR_HANDS) {
		overflowSplit(hand1, upcard);
		return;
	}

	// Allow splitting Aces once and hit each Ace once without playthrough.
	// TODO: look up whether double-down allowed on split hand
	bool playThrough = true;

	if (isPair(hand1) && hand1->cards[0].rank == ACE)
		playThrough = false;

	// Make the new hand
	Hand newHand = Hand_(player);

	// Get card from 1st hand
	Card card = hand1->cards[1];
	hand1->size--;

	// Hit the 1st hand with new card from the deck
	hit(hand1);

	// Hit 2nd hand with card from 1st hand and the deck
	hit(&newHand, &card);

	hit(&newHand);

	// Add 2nd hand to the player
	int index = add(player, &newHand);

	Hand* hand2 = &player->hands[index];

	// Not playing through after hitting
	if (!playThrough)
		return;

	// Play through each hand recursively
	playout(hand1, upcard);

	playout(hand2, upcard);
}

__device__ void overflowSplit(Hand* hand, Card* upcard) {
	Player* player = (Player*)hand->player;

	Strategy* strategy = player->strategy;

	Play play = NO_PLAY;

	if (hand->value >= 12) {
		play = doSection1(hand, upcard, strategy);
	}
	else {
		play = doSection2(hand, upcard, strategy);
	}

	switch (play) {
	case STAY:
		break;

	case HIT:
		hit(hand);

		if (!isBroke(hand))
			playout(hand, upcard);

		break;

	case DOUBLE_DOWN:
		hit(hand);

		hand->bet *= 2.0;

		break;

	case SPLIT:
		// A split here is tantamount to STAY
		break;

	case NO_PLAY:
		//assert(false);
		break;
	}

	return;
}

__device__ void play(Hand* dealer, Player* player, Game* statistics) {
	int remaining = player->size;

	// Payout the hands we can at this point...
	for (int index = 0; index < player->size; index++) {
		Hand* hand = &player->hands[index];

		if (isBroke(hand)) {
			player->pl -= hand->bet;

			statistics->count[BUSTS]++;

			remaining--;
		}
		// A+10 on split hand not "natural" blackjack and doesn't receive bonus.
		// See https://en.wikipedia.org/wiki/Aces_and_eights_(blackjack).
		else if (isBlackjack(player, hand)) {
			player->pl += (hand->bet * PAYOFF_BLACKJACK);

			statistics->count[BLACKJACKS]++;

			remaining--;
		}
		/*
		else if (isCharlie(hand)) {
		player->pl += (hand->bet * PAYOFF_CHARLIE);

		statistics->count[CHARLIES]++;

		remaining--;
		}
		*/
	}

	// If no hands remaining, the dealer does not have to play
	if (remaining == 0)
		return;

	// Dealer stands on (soft or otherwise) 17 or higher 
	while (dealer->value < 17) {
		hit(dealer);
	}

	// Test all the remaining hands
	for (int index = 0; index < player->size; index++) {
		Hand* hand = &player->hands[index];

		// We've handle these above
		if (isBroke(hand) || isBlackjack(player, hand))
			continue;

		// Dealer blackjack beats all except player blackjack and charlie
		if (isBlackjack(dealer)) {
			player->pl -= hand->bet;
			statistics->count[DEALER_BLACKJACKS]++;
		}

		// If dealer broke, pay the player
		else if (isBroke(dealer)) {
			player->pl += hand->bet;
			statistics->count[WINS]++;
		}

		// If dealer lost, pay the player
		else if (dealer->value < hand->value) {
			player->pl += hand->bet;
			statistics->count[WINS]++;
		}

		// If player lost, collect for house
		else if (dealer->value > hand->value) {
			player->pl -= hand->bet;
			statistics->count[LOSSES]++;
		}

		// If hands same, nobody wins or loses
		else if (dealer->value == hand->value) {
			player->pl += 0;
			statistics->count[PUSHES]++;
		}
		else
			/* DO NOTHING */ ;
	}
}

__device__ Play getPlay(Hand* hand, Card* upcard) {
	Strategy* strategy = ((Player*)hand->player)->strategy;

	Play play1 = NO_PLAY;

	if (isPair(hand))
		play1 = doSection4(hand, upcard, strategy);

	else if (isAcePlusX(hand))
		play1 = doSection3(hand, upcard, strategy);

	else if (hand->value >= 5 && hand->value <= 11)
		play1 = doSection2(hand, upcard, strategy);

	else
		play1 = doSection1(hand, upcard, strategy);

	return play1;

	// Commented code below left for debugging only.
/*
	int dealer = isFace(upcard) ? 10 : upcard->rank;

	if (dealer == ACE)
		dealer += ACE_AS_11;

	Play play2 = NO_PLAY;

	if (isPair(hand) && (hand->cards[0].rank == ACE || hand->cards[0].rank == EIGHT))
		play2 = SPLIT;

	else if (hand->value == 11 && hand->size == 2)
		play2 = DOUBLE_DOWN;

	else if (hand->value <= 10)
		play2 = HIT;

	else if (hand->value >= 17)
		play2 = STAY;

	else if (hand->value <= 16 && dealer <= 6)
		play2 = STAY;

	else if (hand->value <= 16 && dealer > 6)
		play2 = HIT;

	//else
		//assert(false);

	return play2;
*/
}

__device__ Game go(int ngames, Strategy* strategy) {
	Game statistics = Game_();

	for (int gameno = 0; gameno < ngames; gameno++) {
		play(strategy, &statistics);
	}
	
	return statistics;
}


/////////////// See http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states_) {
	// Calculate the thread's index
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	/* we have to initialize the state */
	hiprand_init(seed, // the seed can be the same for each core, here we pass the time in from the CPU
		index,        // the sequence number should be different for each thread (unless you want all
					  // thread to get the same sequence of numbers for some reason - use thread id!
		0,            // the offset is how much extra we advance in the sequence for each call, can be 0
		&states_[index]);

	states = states_;
}

/* this GPU kernel takes plays n games given pointer to the strategies and pointer return statistics. */
__global__ void run(unsigned int numGames, Strategy* strategies, Game* statistics) {
	// Calculate the thread's unique index
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	// Play the games
	statistics[index] = go(numGames, &strategies[index]);
}

/////////////// GPU driver

int evaluate(int numThreads, Strategy* strategies, int numGames, Game* statistics, int seed) {
	return evaluate(1, numThreads, strategies, numGames, statistics, seed);
}

 int evaluate(int numBlocks, int numThreads, Strategy* strategies, int numGames, Game* statistics, int seed) {
	try {
//		printf("start evaluate function\n");

		// Setup and initialize the device

		// Choose the GPU to run on
		hipError_t cudaStatus = hipSetDevice(0);
		check(cudaStatus, "could not find gpu device!");
//		printf("hipSetDevice: %d\n",cudaStatus);
		
		
		// Allocate a stack space per thread on the GPU
		cudaStatus = hipDeviceSetLimit(hipLimitStackSize, LIMIT_STACK_SIZE);
		check(cudaStatus, "set stack limit failed!");
//		printf("hipDeviceSetLimit: %d\n",cudaStatus);

		size_t asize;
		cudaStatus = hipDeviceGetLimit(&asize, hipLimitStackSize);
		check(cudaStatus, "get stack limit test failed!");
//		printf("hipDeviceGetLimit: %d asize=%d\n",cudaStatus,asize);

		if (asize != LIMIT_STACK_SIZE)
			check(hipErrorNotSupported, "test stack limit failed!");

		// Random states per thread
		hiprandState_t* dev_states;

		// Allocate space on the GPU for the random states
		cudaStatus = hipMalloc((void**)&dev_states, numThreads * sizeof(hiprandState_t));
		check(cudaStatus, "failed to malloc random states!");
//		printf("hipMalloc: %d\n",cudaStatus);

		// Invoke kernel to initialize all of the random states
		init <<< numBlocks, numThreads/numBlocks >>> (seed, dev_states);
//		printf("init random states done!\n");		

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		check(cudaStatus, "failed to launch kernel to initialize random states on device!");

		cudaStatus = hipDeviceSynchronize();
		check(cudaStatus, "failed to initialize random states on device!");

		// Create the strategies
		Strategy* dev_strategies = 0;

		// Allocate the space on the GPU for the strategies

		cudaStatus = hipMalloc((void**)&dev_strategies, numThreads * sizeof(Strategy));
		check(cudaStatus, "failed to malloc stategies on device!");

		// Copy the strategies to the GPU
		cudaStatus = hipMemcpy(dev_strategies, strategies, numThreads * sizeof(Strategy), hipMemcpyHostToDevice);
//		printf("hipMemcpy strategies to device: %d\n",cudaStatus);
		
		check(cudaStatus, "failed to copy stategies to device!");

		// Allocate space on the GPU for the game statistics
		Game* dev_statistics = 0;

		cudaStatus = hipMalloc((void**)&dev_statistics, numThreads * sizeof(Game));
		check(cudaStatus, "failed to malloc statistics on device!");
//		printf("hipMalloc strategies %d bytes: %d\n",cudaStatus,numThreads * sizeof(Game));

		// There is no need to copy the statistics to the GPU since they are generated by the GPU

		// Finally invoke the kernel to run the games */
		run <<< numBlocks, numThreads/numBlocks >>> (numGames, dev_strategies, dev_statistics);
//		printf("run %d games returned!\n",numGames);		
		
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		check(cudaStatus, "failed to launch kernel to run games on device!");
//		printf("hipGetLastError: %d\n",cudaStatus);


		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		check(cudaStatus, "failed to run games on device!");
//		printf("hipDeviceSynchronize: %d\n",cudaStatus); 

		// Copy results back to host
		cudaStatus = hipMemcpy(strategies, dev_strategies, numThreads * sizeof(Strategy), hipMemcpyDeviceToHost);
//		printf("hipMemcpy results %d bytes: %d\n",cudaStatus,numThreads * sizeof(Strategy));

		check(cudaStatus, "failed to copy back strategies from device!");
//		printf("hipMemcpy results %d bytes: %d\n",cudaStatus,numThreads * sizeof(Strategy));

		hipMemcpy(statistics, dev_statistics, numThreads * sizeof(Game), hipMemcpyDeviceToHost);
		check(cudaStatus, "failed to copy back statistics from device!");
//		printf("hipMemcpy statistics %d bytes: %d\n",cudaStatus,numThreads * sizeof(Game));

		/* free the memory we allocated for the states and numbers */
		hipFree(dev_states);
		// We don't use dev_games
		//hipFree(dev_games);
		hipFree(dev_strategies);
		hipFree(dev_statistics);

		return OK;
	}
	catch (hipError_t err) {
		return err;
	}
}


